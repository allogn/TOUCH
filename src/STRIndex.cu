#include "hip/hip_runtime.h"
#include "STRIndex.h"

STRIndex::STRIndex()
{
        objectCount=0;
        objectSize=0;
        pageCount=0;
        binCount=0;
        objectPerPage=0;
        objectPerXBins=0;
        objectPerYBins=0;
        payload = new FLAT::PayLoad();
        metadataStructure = new vector<FLAT::MetadataEntry*>();
}

STRIndex::~STRIndex()
{
        delete payload;
        delete metadataStructure;
}

void STRIndex::buildIndex(FLAT::SpatialObjectStream* input,string indexFileStem)
{
#ifdef PROFILING
        FLAT::Timer tesselation,seeding,linker;
        tesselation.start();
#endif
#ifdef INFORMATION
        cout << "\n == STR TESSELLATION ==\n\n";
#endif
        initialize(input,indexFileStem);
        doTessellation(input);
#ifdef PROFILING
        tesselation.stop();
        cout << "Tessellation Duration: " << tesselation << "\n";
        linker.start();
#endif
#ifdef INFORMATION
        cout << "\n == LINKER RTREE BUILDING ==\n\n";
#endif
        FLAT::MetaDataStream* metaStream = new FLAT::MetaDataStream(metadataStructure);
        SpatialIndex::IStorageManager* rtreeStorageManager = SpatialIndex::StorageManager::createNewMemoryStorageManager();
        FLAT::uint32 fanout = (FLAT::uint32)floor(PAGE_SIZE-76+0.0)/(objectSize+12+0.0);

        SpatialIndex::id_type indexIdentifier=1;
        SpatialIndex::ISpatialIndex *linkerTree = SpatialIndex::RTree::createAndBulkLoadNewRTree (
                SpatialIndex::RTree::BLM_STR,
                *metaStream,
                *rtreeStorageManager,
                        0.9999, fanout,
                        fanout, DIMENSION,
                SpatialIndex::RTree::RV_RSTAR,
                indexIdentifier);
#ifdef PROFILING
        linker.stop();
        cout << "Linker Creation Duration: " << linker << "\n";
        seeding.start();
#endif
#ifdef INFORMATION
        cout << "\n == BUILDING SEED INDEX WHILE INDUCING LINKS ==\n\n";
#endif
        FLAT::MetaDataStream* metaDataStream = new FLAT::MetaDataStream(metadataStructure,linkerTree);

        FLAT::SeedBuilder::buildSeedTree(indexFileStem,metaDataStream);

#ifdef DEBUG
        cout << "TOTAL PAGES: " << metaDataStream->pages <<endl;
        cout << "TOTAL LINKS ADDED: " << metaDataStream->links <<endl;
        cout << "SUMMED VOLUME: " << metaDataStream->sumVolume <<endl;
        cout << "AVERAGE LINKS: " << ((metaDataStream->links+0.0) / (metaDataStream->pages+0.0)) << endl;
        cout << "AVERAGE VOLUME: " << ((metaDataStream->sumVolume+0.0) / (metaDataStream->pages+0.0)) << endl;

        for (int i=0;i<100;i++)
                cout << metaDataStream->frequency[i] << "\n";

        //cout << "OVERFLOW VOLUME: " << metaDataStream->overflow <<endl;
        //for (int i=0;i<100;i++)
        //	cout << metaDataStream->volumeDistributon[i] << "\t" << metaDataStream->volumeLink[i] << "\t"
                 //    << ( (metaDataStream->volumeLink[i]+0.0)/(metaDataStream->volumeDistributon[i]+0.0)) << "\n" ;
#endif
        delete metaDataStream;
#ifdef PROFILING
        seeding.stop();
        cout << "Building Seed Structure & Links Duration: " << seeding << "\n";
#endif
}

void STRIndex::initialize(FLAT::SpatialObjectStream* input,string indexFileStem)
{
        objectCount     = input->objectCount;
        objectSize      = input->objectByteSize;
        objectType		= input->objectType;
        universe        = input->universe;
        //FLAT::Box::infiniteFLAT::Box(universe); // to make open ended Partition MBRs in the corner of universe... dont do it

#ifdef BBP
        objectPerPage = (FLAT::uint64)floor((PAGE_SIZE-4.0) / (objectSize+0.0)); // minus 4 bytes because each page has an int counter with it
#else
        objectPerPage = (FLAT::uint64)floor((PAGE_SIZE-4.0) / (objectSize+0.0));
        //objectPerPage   = 67; // to Degrade FLAT to change back remove comment
#endif
        pageCount       = (FLAT::uint64)ceil( (objectCount+0.0) / (objectPerPage+0.0) );
        binCount        = pow (pageCount,1.0/(3+0.0));

        objectPerXBins  = (FLAT::uint64)ceil((objectCount+0.0) / binCount);
        objectPerYBins  = (FLAT::uint64)ceil((objectPerXBins+0.0) / binCount);

#ifdef DEBUG
                cout << "MINIMUM PAGES NEED TO STORE DATA: "<<pageCount <<endl
                 << "PAGES BINS PER DIMENSION: " << binCount << endl
                 << "OBJECTS IN EVERY X BIN: " << objectPerXBins << endl
                 << "OBJECTS IN EVERY Y BIN: " << objectPerYBins << endl
                 << "OBJECTS IN EVERY Z BIN or PAGE: " << objectPerPage << endl;
#endif
        metadataStructure->reserve(pageCount);
        payload->create(indexFileStem,PAGE_SIZE,objectPerPage,objectSize,objectType);
}

void STRIndex::doTessellation(FLAT::SpatialObjectStream* input)
{
        FLAT::ExternalSort* xSort = new FLAT::ExternalSort(SORTING_FOOTPRINT_MB,0,objectType);
        FLAT::ExternalSort* ySort = new FLAT::ExternalSort(SORTING_FOOTPRINT_MB,1,objectType);
        FLAT::ExternalSort* zSort = new FLAT::ExternalSort(SORTING_FOOTPRINT_MB,2,objectType);
        FLAT::Box Partition = universe;
        FLAT::uint64 PageCount=0;
        FLAT::uint64 dataCount=0;

#ifdef DEBUG
        int idx=0,idy=0,idz=0;
#endif
        // SORTING AND BINNING
#ifdef INFORMATION
        cout << "READING INPUT...." << endl;
#endif
        while (input->hasNext())
                xSort->insert(input->getNext());
#ifdef INFORMATION
        cout << "\nSORTING...." << endl;
#endif
        xSort->sort();

        FLAT::uint64 xCount=0,oldCountX=0;

        while(xSort->hasNext())
        {
                FLAT::SpatialObject* xtemp = xSort->getNext();
                ySort->insert(xtemp);
                xCount++;
                if (xCount%objectPerXBins==0 || xCount==objectCount)
                {
                        if (xCount>objectPerXBins) Partition.low[0] = Partition.high[0];
                        if (xCount==objectCount) Partition.high[0] = universe.high[0];
                        else Partition.high[0] = xtemp->getSortDimension(0);
                        ySort->sort();
                        FLAT::uint64 yCount=0,oldCountY=0;

                        while(ySort->hasNext())
                        {
                                FLAT::SpatialObject* ytemp = ySort->getNext();
                                zSort->insert(ytemp);
                                yCount++;

                                if (yCount%objectPerYBins==0 || yCount==xCount-oldCountX)
                                {
                                        if (yCount>objectPerYBins) Partition.low[1] = Partition.high[1];
                                        if (yCount==xCount-oldCountX) Partition.high[1] = universe.high[1];
                                        else Partition.high[1] = ytemp->getSortDimension(1);
                                        zSort->sort();

                                        /////////////////// MAKING META AND PAYLOAD PAGES ////////////////////
                                        vector<FLAT::SpatialObject*> items;
                                        FLAT::Box PageMBR;
                                        FLAT::uint64 zCount=0;

                                        while (zSort->hasNext())
                                        {
                                                FLAT::SpatialObject* temp = zSort->getNext();
                                                //cout << ((Cone*)temp) <<" - " << temp->getSortDimension(0) << " - " << temp->getSortDimension(1) << " - "<< temp->getSortDimension(2) <<endl;
                                                items.push_back(temp);
                                                zCount++;
                                                dataCount++;
#ifdef PROGRESS
                                                if (dataCount%10000000==0) cout << "INDEXING OBJECTS: " << dataCount << " DONE"<< endl;
#endif
                                                if (zCount%objectPerPage==0 || zCount==yCount-oldCountY)
                                                {
                                                        FLAT::Box::boundingBox(PageMBR,items);
                                                        if (zCount>objectPerPage) Partition.low[2] = Partition.high[2];
                                                        if (zCount==yCount-oldCountY) Partition.high[2] = universe.high[2];
                                                        else Partition.high[2] = temp->getSortDimension(2);
                                                        FLAT::MetadataEntry* metaEntry = new FLAT::MetadataEntry();
                                                        metaEntry->pageMbr = PageMBR;
                                                        metaEntry->partitionMbr = Partition + PageMBR;
                                                        metaEntry->pageId = PageCount;
#ifdef DEBUG
                                                        metaEntry->i = idx; metaEntry->j = idy; metaEntry->k =idz;
                                                        //cout <<metaEntry->pageId << " ["<< idx << "," << idy << "," << idz << "] \t" << metaEntry->partitionMbr  << "\t" << metaEntry->pageMbr<< endl;
#endif
                                                        metadataStructure->push_back(metaEntry);

                                                        PageCount++;
                                                        payload->putPage(items);
                                                        items.clear();
                                                        if (zCount==yCount-oldCountY) break;
#ifdef DEBUG
                                                        idz++;
#endif
                                                }

                                        }
                                        ////////////////////////////////////////////////////////////////////
                                        Partition.low[2] = universe.low[2];
                                        zSort->clean();
                                        oldCountY = yCount;
                                        if (yCount==xCount-oldCountX) break;
#ifdef DEBUG
                                        idy++;idz=0;
#endif
                                }
                        }
                        Partition.low[1] = universe.low[1];
                        ySort->clean();
                        oldCountX = xCount;
                        if (xCount==objectCount) break;
#ifdef DEBUG
                        idx++;idy=0;idz=0;
#endif
                }
        }
        Partition.low[0] = universe.low[0];
        xSort->clean();

#ifdef INFORMATION
        cout << "PAGES USED FOR INDEX: " << PageCount << endl;
        cout << "OBJECTS INDEXED: " << dataCount << endl;
#endif
        delete zSort;
        delete ySort;
        delete xSort;
}

void STRIndex::induceConnectivityFaster()
{
        uint32_t pages = metadataStructure->size();
        uint32_t hopFactor = (uint32_t)((floor( (objectPerXBins+0.0) / (objectPerYBins+0.0)) *
                                      ceil ( (objectPerYBins+0.0) / (objectPerPage+0.0))) +
                                      ceil ( ((objectPerXBins%objectPerYBins)+0.0) / (objectPerPage+0.0)));
#ifdef DEBUG
        uint32_t links=0;
#endif
        for (uint32_t i=0;i<pages;++i)
        {
                for (uint32_t j=i+1;j<pages;++j)
                {
                        if (metadataStructure->at(i)->partitionMbr.high[0] < metadataStructure->at(j)->partitionMbr.low[0])
                                break;

                        if ((metadataStructure->at(i)->partitionMbr.high[1] < metadataStructure->at(j)->partitionMbr.low[1]))
                        {
                                uint32_t nextHop = ((j/hopFactor)+1)*hopFactor;
                                if (nextHop < pages)
                                        j = nextHop;
                        }

                        if (FLAT::Box::overlap( metadataStructure->at(i)->partitionMbr , metadataStructure->at(j)->partitionMbr ))
                        {
                                metadataStructure->at(i)->pageLinks.insert(j);
                                metadataStructure->at(j)->pageLinks.insert(i);
#ifdef DEBUG
                                links+=2;
#endif
                        }
                }
#ifdef PROGRESS
                if (i%100000==0 && i>0) cout << "INDUCING LINKS: "<< i << " PAGES DONE" << endl;
#endif
        }

#ifdef DEBUG
        cout << "TOTAL PAGES: " << pages <<endl;
        cout << "TOTAL LINKS ADDED: "<< links <<endl;

//		int frequency[100];
//		for (int i=0;i<100;i++) frequency[i]=0;
//		for (uint32_t j=0;j<pages;j++)
//		{
//			if (metadataStructure->at(j)->pageLinks.size()>100)
//			{
//				cout << "id("<< j<< ") = [" <<  metadataStructure->at(j)->i << "," << metadataStructure->at(j)->j << "," << metadataStructure->at(j)->k << "] \tLINKS:" << metadataStructure->at(j)->pageLinks.size() << " \tMBR" << metadataStructure->at(j)->partitionMbr << "\n";
//
//				//for (set<id>::iterator i = metadataStructure->at(j)->pageLinks.begin();i !=  metadataStructure->at(j)->pageLinks.end(); ++i)
//				//	cout << "\tid("<< *i << ") = [" <<  metadataStructure->at(*i)->i << "," << metadataStructure->at(*i)->j << "," << metadataStructure->at(*i)->k << "] \tLINKS:" << metadataStructure->at(*i)->pageLinks.size() << " \tMBR" << metadataStructure->at(*i)->partitionMbr << "\n";
//			}
//			if (metadataStructure->at(j)->pageLinks.size()<100)
//				frequency[metadataStructure->at(j)->pageLinks.size()]++;
//		}


//		for (int i=0;i<100;i++)
//			cout << "Links: " << i << " Frequency: " << frequency[i] << "\n";
#endif
}

void STRIndex::loadIndex(string indexFileStem)
{
        payload->load(indexFileStem);
        vector<FLAT::SpatialObject*> items;
        for (int i=0;i<41;i++)
        {
                payload->getPage(items,i);
                items.clear();
        }
}

